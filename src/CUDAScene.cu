#include "hip/hip_runtime.h"
#include "CUDAScene.cuh"

__host__ CUDAScene::CUDAScene()
{
	visibles = NULL;

	materials = NULL;

	index_arrays = NULL;

	vertex_arrays = NULL;
}

__host__ CUDAScene::CUDAScene(UnifiedArray<CUDAVisible*>* const visibles, UnifiedArray<Material<CUDA_RNG>*>* const materials)
	//: visibles(visibles), materials(materials)
{
	this->visibles = visibles;
	this->materials = materials;

	index_arrays = NULL;

	vertex_arrays = NULL;

}

CUDAScene::CUDAScene(const unsigned int visible_count, const unsigned int material_count)
{

	visibles = new UnifiedArray<CUDAVisible*>(visible_count);

	materials = new UnifiedArray<Material<CUDA_RNG>*>(material_count);

}

__host__ CUDAScene::CUDAScene(const std::string& fp)
{
	std::ifstream input_file(fp);
	nlohmann::json j;
	input_file >> j;

	unsigned int visible_count = j.size();

	visibles = new UnifiedArray<CUDAVisible*>(visible_count);

	materials = new UnifiedArray<Material<CUDA_RNG>*>(visible_count);

	UnifiedArray<CUDASphere>* host_spheres = new UnifiedArray<CUDASphere>(visible_count);

	for (unsigned int i = 0; i < visible_count; i++)
	{
		auto json_visible = j[i];

		if (json_visible["type"] == "Sphere")
		{

			vec3 center = vec3(json_visible["center"][0], json_visible["center"][1], json_visible["center"][2]);

			(*host_spheres)[i] = CUDASphere(center, (float)json_visible["radius"], NULL);

			auto json_material = json_visible["material"];

			vec3 albedo = vec3(json_material["albedo"][0], json_material["albedo"][1], json_material["albedo"][2]);

			(*materials)[i] = Material<CUDA_RNG>(
				albedo,
				json_material["diffuse"],
				json_material["metal"],
				json_material["dielectric"],
				json_material["roughness"],
				json_material["refractive_index"]
			).to_device();
		}
	}


	checkCudaErrors(hipDeviceSynchronize());

	int threads = 512;

	int blocks = visible_count / threads + 1;

	instantiate_spheres << <blocks, threads >> > (this, host_spheres);

	checkCudaErrors(hipDeviceSynchronize());

	delete host_spheres;
}

__global__ void instantiate_spheres(CUDAScene* const scene, const UnifiedArray<CUDASphere>* const spheres)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	if (id < spheres->size())
	{
		(*scene->visibles)[id] = new CUDASphere(
			(*spheres)[id].center,
			(*spheres)[id].radius,
			(*scene->materials)[id]
		);
	}
}

/*
__device__ CUDAScene::CUDAScene(const CUDAScene& cs)
{
	visibles = cs.visibles;
	materials = cs.materials;
	index_arrays = cs.index_arrays;
	vertex_arrays = cs.vertex_arrays;
}

__device__ CUDAScene& CUDAScene::operator=(const CUDAScene& cs)
{
	if (this == &cs)
		return *this;

	set_visibles(cs.visibles);

	set_materials(cs.materials);

	return *this;
}

__device__ CUDAScene::CUDAScene(CUDAScene&& cs)
{

	set_visibles(cs.visibles);

	cs.visibles = NULL;

	set_materials(cs.materials);

	cs.materials = NULL;

}

// Move assignment
__device__ CUDAScene& CUDAScene::operator=(CUDAScene&& cs) 
{

	if (this == &cs)
		return *this;

	set_visibles(cs.visibles);

	cs.visibles = NULL;

	set_materials(cs.materials);

	cs.materials = NULL;

	return *this;
}
*/

__host__ CUDAScene::~CUDAScene()
{

	delete_visibles();

	delete_materials();

	delete_vertex_arrays();

	delete_index_arrays();

}

__host__ void CUDAScene::set_visibles(UnifiedArray<CUDAVisible*>* const new_visibles)
{
	delete_visibles();

	visibles = new_visibles;
}

__host__ void CUDAScene::set_materials(UnifiedArray<Material<CUDA_RNG>*>* const new_materials)
{

	delete_materials();

	materials = new_materials;
}

__global__ void cuda_delete_visibles(UnifiedArray<CUDAVisible*>* visibles)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	if (id < visibles->size())
		delete (*visibles)[id];

}

__host__ void CUDAScene::delete_visibles()
{
	int threads = 512;

	int blocks = visibles->size() / threads + 1;

	if (visibles)
		cuda_delete_visibles<<<blocks, threads>>>(visibles);


	checkCudaErrors(hipDeviceSynchronize());

	delete visibles;

}

__host__ void CUDAScene::delete_materials()
{
	if (materials)
	{
		for (uint32_t i = 0; i < materials->size(); i++)
			hipFree((*materials)[i]);

		hipFree(materials);
	}
}


__host__ void CUDAScene::delete_vertex_arrays()
{
	if (vertex_arrays)
	{

		for (int i = 0; i < vertex_arrays->size(); i++)
			hipFree((*vertex_arrays)[i]);

		hipFree(vertex_arrays);
	}

}

__host__ void CUDAScene::delete_index_arrays()
{
	if (index_arrays)
	{
		for (int i = 0; i < index_arrays->size(); i++)
			hipFree((*index_arrays)[i]);

		hipFree(index_arrays);
	}

}

