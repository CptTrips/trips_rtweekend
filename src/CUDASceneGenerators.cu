#include "hip/hip_runtime.h"
#include "CUDASceneGenerators.cuh"


CUDAScene* scene_factory(const int visible_count, const int material_count)
{

	CUDAScene* scene = new CUDAScene();

	UnifiedArray<CUDAVisible*>* visibles = new UnifiedArray<CUDAVisible*>(visible_count);

	UnifiedArray<Material<CUDA_RNG>*>* materials = new UnifiedArray<Material<CUDA_RNG>*>(material_count);

	scene->visibles = visibles;

	scene->materials = materials;

	return scene;
}


CUDAScene* rtweekend(int attempts, int seed)
{
	CPU_RNG rng = CPU_RNG(seed);

	std::vector<Material<CUDA_RNG>*> materials;

	std::vector<vec3> centers;

	for (int a = -attempts/2; a < attempts/2; a++)
	{
		for (int b = -attempts; b < attempts; b++)
		{

			float material_coin = rng.sample();

			vec3 center(a + 0.9f * rng.sample(), 0.2, b + 0.9 * rng.sample());

			if ((center - vec3(4, 0.2, 0)).length() > 0.9)
			{
				Material<CUDA_RNG>* mat;

				if (material_coin < 0.8)
				{
					vec3 albedo(rng.sample() * rng.sample(), rng.sample() * rng.sample(), rng.sample() * rng.sample());

					mat = new Diffuse<CUDA_RNG>(albedo);
				}
				else if (material_coin < 0.95)
				{
					vec3 albedo(0.5 * (1 + rng.sample()), 0.5 * (1 + rng.sample()), 0.5 * (1 + rng.sample()));

					float roughness = rng.sample();

					mat = new Metal<CUDA_RNG>(albedo, roughness);
				}
				else
				{
					vec3 albedo(1, 1, 1);

					mat = new Dielectric<CUDA_RNG>(albedo, 1.5);
				}

				materials.push_back(mat);
				centers.push_back(center);
			}
		}
	}

	unsigned int random_sphere_count = materials.size();

	CUDAScene* scene = new CUDAScene(random_sphere_count + 4, random_sphere_count + 4);

	UnifiedArray<vec3>* device_centers = new UnifiedArray<vec3>(random_sphere_count);

	for (unsigned int i = 0; i < random_sphere_count; i++)
	{
		(*scene->materials)[i] = materials[i]->to_device();

		checkCudaErrors(hipDeviceSynchronize());

		delete materials[i];

		(*device_centers)[i] = centers[i];
	}

	materials.clear();

	Material<CUDA_RNG>* ground_mat = new Diffuse<CUDA_RNG>(vec3(0.5, 0.5, 0.5));
	
	Material<CUDA_RNG>* dielectric_mat = new Dielectric<CUDA_RNG>(vec3(1., 1., 1.), 1.5);

	Material<CUDA_RNG>* diffuse_mat = new Diffuse<CUDA_RNG>(vec3(0.4, 0.2, 0.1));

	Material<CUDA_RNG>* metal_mat = new Metal<CUDA_RNG>(vec3(0.7, 0.6, 0.5), 0.);

	(*scene->materials)[random_sphere_count] = ground_mat->to_device();
	(*scene->materials)[random_sphere_count + 1] = dielectric_mat->to_device();
	(*scene->materials)[random_sphere_count + 2] = diffuse_mat->to_device();
	(*scene->materials)[random_sphere_count + 3] = metal_mat->to_device();

	delete ground_mat;
	delete dielectric_mat;
	delete diffuse_mat;
	delete metal_mat;


	int threads = 1;

	int blocks = 1;

	gen_rtweekend << <blocks, threads >> > (scene, device_centers);

	checkCudaErrors(hipDeviceSynchronize());

	return scene;
}


__global__ void gen_rtweekend(CUDAScene* scene, UnifiedArray<vec3>* device_centers)
{
	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id == 0)
	{
		unsigned int visibles_count = scene->visibles->size();
		for (unsigned int i = 0; i < visibles_count - 4; i++)
		{
			(*scene->visibles)[i] = new CUDASphere((*device_centers)[i], 0.2f, (*scene->materials)[i]);
		}

		// ground
		(*scene->visibles)[visibles_count - 4] = new CUDASphere(vec3(0, -1000, 0), 1000, (*scene->materials)[visibles_count - 4]);

		// dielectric
		(*scene->visibles)[visibles_count - 3] = new CUDASphere(vec3(0, 1, 0), 1.0, (*scene->materials)[visibles_count - 3]);

		// diffuse
		(*scene->visibles)[visibles_count - 2] = new CUDASphere(vec3(-4, 1, 0), 1.0, (*scene->materials)[visibles_count - 2]);

		// metal
		(*scene->visibles)[visibles_count - 1] = new CUDASphere(vec3(4, 1, 0), 1.0, (*scene->materials)[visibles_count - 1]);
	}
}

CUDAScene* random_balls(const int ball_count)
{

	CUDAScene* scenery = new CUDAScene(ball_count, 2);

	for (unsigned int i = 0; i < ball_count; i++)
		(*scenery->materials)[i] = Material<CUDA_RNG>().to_device();

	int threads = 512;

	int blocks = ball_count / threads + 1;

	gen_random_balls << <blocks, threads >> > (scenery, ball_count);

	checkCudaErrors(hipDeviceSynchronize());

	return scenery;
}


__global__ void gen_random_balls(CUDAScene* const scene, const int ball_count)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;


	if (id < ball_count)
	{

		CUDA_RNG rng = CUDA_RNG(my_cuda_seed, id);

		float r = 0.33; // ball radius

		float xrange = 6.f;
		float yrange = 3.75;
		float zrange = 2.5f;

		float zoffset = -0.f;

		vec3 center = vec3(
			xrange * (2.f * rng.sample() - 1)
			,yrange * (2.f * rng.sample() - 1)
			,zoffset - zrange * rng.sample()
		);

		vec3 color = vec3(rng.sample(),rng.sample(),rng.sample());

		float roughness = 3.f*rng.sample();

		// Randomize the material
		Material<CUDA_RNG>* m = (*scene->materials)[id];

		if (rng.sample() > .5f) {

			*m = Metal<CUDA_RNG>(color, roughness);

		} else {

			*m = Diffuse<CUDA_RNG>(color);

		}

		(*scene->visibles)[id] = new CUDASphere(center, r, m);

	}

}


CUDAScene* single_ball()
{
	CUDAScene* scenery = scene_factory(1, 1);

	gen_single_ball << <1, 1>> > (scenery);

	checkCudaErrors(hipDeviceSynchronize());

	return scenery;
}


__global__ void gen_single_ball(CUDAScene* const scene)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id < 1)
	{
		vec3 center = vec3(3.f, 0.f, 0.f);
		float radius = 1.f;
		Material<CUDA_RNG>* mat = new Diffuse<CUDA_RNG>(vec3(1.f, 0.f, 0.f));

		(*scene->visibles)[id] = new CUDASphere(center, radius, mat);
		(*scene->materials)[id] = mat;
	}
}



CUDAScene* single_triangle()
{

	CUDAScene* scenery = scene_factory(1, 1);

	gen_single_triangle << <1, 1 >> > (scenery);

	checkCudaErrors(hipDeviceSynchronize());

	return scenery;
}


__global__ void gen_single_triangle(CUDAScene* const scene)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id == 0)
	{
		vec3 a = vec3(0.f, 0.f, 1.f);

		vec3 b = vec3(0.f, 1.f, 1.f);

		vec3 c = vec3(1.f, 0.f, 1.f);

		vec3 points[3] = { a, b, c };

		Material<CUDA_RNG>* mat = new Metal<CUDA_RNG>(vec3(.5f, .2f, .2f), 0.1f);

		(*scene->visibles)[id] = new Triangle(points, mat);
		(*scene->materials)[id] = mat;

	}
}


Array<vec3>* cube_vertices(const vec3& translation = vec3(0.f, 0.f, 0.f))
{

	Array<vec3>* vertex_array = new Array<vec3>(8);

	for (int i = 0; i < 2; i++)
		for (int j = 0; j < 2; j++)
			for (int k = 0; k < 2; k++)
			{
				(*vertex_array)[i + 2 * j + 4 * k] = vec3(i, j, k) + translation;
			}

	return vertex_array;
}

Array<uint32_t>* cube_indices()
{
	Array<uint32_t>* index_array = new Array<uint32_t>(36);

	int indices[36] = {
		0, 2, 1, 2, 3, 1,
		1, 5, 4, 0, 1, 4,
		4, 5, 7, 4, 7, 6,
		6, 7, 3, 3, 2, 6,
		1, 3, 5, 3, 7, 5,
		0, 4, 6, 0, 6, 2
	};

	for (int i = 0; i < 36; i++)
		(*index_array)[i] = indices[i];

	return index_array;
}

CUDAScene* single_cube()
{

	Array<vec3>* vertex_array = cube_vertices();

	Array<vec3>* const device_vertex_array = vertex_array->to_device();

	Array<uint32_t>* index_array = cube_indices();

	Array<uint32_t>* const device_index_array = index_array->to_device();

	Material<CUDA_RNG>* mat = new Diffuse<CUDA_RNG>(vec3(0.7f, 0.1f, 0.2f));

	Material<CUDA_RNG>* const device_mat = mat->to_device();

	CUDAScene* scenery = scene_factory(1,1);

	gen_single_cube << <1, 1 >> > (scenery, device_vertex_array, device_index_array, device_mat);

	checkCudaErrors(hipDeviceSynchronize());

	delete vertex_array;
	delete index_array;
	delete mat;

	return scenery;
}


__global__ void gen_single_cube(CUDAScene* const scene, const Array<vec3>* const vertex_array, const Array<uint32_t>* const index_array, Material<CUDA_RNG>* const mat)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id == 0)
	{

		(*scene->visibles)[id] = new Mesh(vertex_array, index_array, mat);
		(*scene->materials)[id] = mat;
	}
}

template<typename T>
__host__ T* move_to_device(T* const obj)
{
	T* device_ptr = obj->to_device();

	delete obj;

	return device_ptr;
}


CUDAScene* n_cubes(const int& n)
{

	UnifiedArray<CUDAVisible*>* visibles = new UnifiedArray<CUDAVisible*>(n);
	UnifiedArray<Array<vec3>*>* vertex_arrays = new UnifiedArray<Array<vec3>*>(n);
	UnifiedArray<Array<uint32_t>*>* index_arrays = new UnifiedArray<Array<uint32_t>*>(n);
	UnifiedArray<Material<CUDA_RNG>*>* material_array = new UnifiedArray<Material<CUDA_RNG>*>(n);

	for (int i = 0; i < n; i++)
	{
		const Array<vec3>* vertex_array = cube_vertices(vec3(0.f, 0.f, 1.5f*i));

		(*vertex_arrays)[i] = vertex_array->to_device();

		delete vertex_array;

		Array<uint32_t>* index_array = cube_indices();

		(*index_arrays)[i] = index_array->to_device();

		delete index_array;

		(*material_array)[i] = Diffuse<CUDA_RNG>(vec3((float)i / (float)(n - 1), .5f, 1.f - (float)i / (float)(n - 1))).to_device();

	}

	CUDAScene* scene = new CUDAScene();

	scene->visibles = visibles;

	scene->materials = material_array;

	scene->vertex_arrays = vertex_arrays;

	scene->index_arrays = index_arrays;

	gen_n_cubes << <1, n >> > (scene);

	checkCudaErrors(hipDeviceSynchronize());

	return scene;

}

__global__ void gen_n_cubes(CUDAScene* const scene)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;

	if (id < scene->visibles->size())
	{

		(*scene->visibles)[id] = new Mesh((*scene->vertex_arrays)[id], (*scene->index_arrays)[id], (*scene->materials)[id]);
	}

}


CUDAScene* triangle_carpet(const unsigned int& n)
{
	CUDAScene* scene = new CUDAScene();

	UnifiedArray<CUDAVisible*>* visibles = new UnifiedArray<CUDAVisible*>(1);
	UnifiedArray<Array<vec3>*>* vertex_arrays = new UnifiedArray<Array<vec3>*>(1);
	UnifiedArray<Array<uint32_t>*>* index_arrays = new UnifiedArray<Array<uint32_t>*>(1);
	UnifiedArray<Material<CUDA_RNG>*>* material_array = new UnifiedArray<Material<CUDA_RNG>*>(1);

	Array<vec3>* vertex_array = new Array<vec3>(n * n);

	Array<uint32_t>* index_array = new Array<uint32_t>(3 * 2 * (n - 1) * (n - 1));

	for (unsigned int i = 0; i < n; i++)
	{
		for (unsigned int j = 0; j < n; j++)
		{
			unsigned int vertex_index = n * i + j;

			(*vertex_array)[vertex_index] = vec3(0.f, 1.f * (float)j / (float)n, 1.f * (float)i / (float)n);

			if ((i < n - 1) && (j < n - 1))
			{
				unsigned int index_index = 6 * (n - 1) * i + 6 * j;
				(*index_array)[index_index] = vertex_index;
				(*index_array)[index_index + 1] = vertex_index + 1;
				(*index_array)[index_index + 2] = vertex_index + n;

				(*index_array)[index_index + 3] = vertex_index + n;
				(*index_array)[index_index + 4] = vertex_index + 1;
				(*index_array)[index_index + 5] = vertex_index + n + 1;

			}
		}
	}


	(*vertex_arrays)[0] = vertex_array->to_device();

	(*index_arrays)[0] = index_array->to_device();

	(*material_array)[0] = Diffuse<CUDA_RNG>(vec3(.5f, .5f, .5f)).to_device();

	checkCudaErrors(hipDeviceSynchronize());

	scene->visibles = visibles;

	scene->materials = material_array;

	scene->vertex_arrays = vertex_arrays;

	scene->index_arrays = index_arrays;

	gen_carpet << <1, 1 >> > (scene);

	checkCudaErrors(hipDeviceSynchronize());

	delete vertex_array;

	delete index_array;

	return scene;
}


__global__ void gen_carpet(CUDAScene* const scene)
{
	unsigned int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id == 0)
	{
		(*scene->visibles)[id] = new Mesh((*scene->vertex_arrays)[0], (*scene->index_arrays)[0], (*scene->materials)[0]);
	}
}



