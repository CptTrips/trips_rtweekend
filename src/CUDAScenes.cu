#include "hip/hip_runtime.h"
#include "CUDAScenes.cuh"


CUDAVisible** random_balls(const int ball_count)
{

    CUDAVisible** scenery;

	hipMalloc(&scenery, ball_count * sizeof(scenery));

	int threads = 512;

	int blocks = ball_count / threads + 1;

	gen_random_balls << <blocks, threads >> > (scenery, ball_count);

	hipDeviceSynchronize();

	return scenery;
}

__global__ void gen_random_balls(CUDAVisible** const scenery, const int ball_count)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;


	if (id < ball_count)
	{

		CUDA_RNG rng = CUDA_RNG(my_cuda_seed, id);

		float r = 0.33; // ball radius

		float xrange = 6.f;
		float yrange = 3.75;
		float zrange = 2.5f;

		float zoffset = -0.f;

		vec3 center = vec3(
			xrange * (2.f * rng.sample() - 1)
			,yrange * (2.f * rng.sample() - 1)
			,zoffset - zrange * rng.sample()
		);

		vec3 color = vec3(rng.sample(),rng.sample(),rng.sample());

		float roughness = 3.f*rng.sample();

		// Randomize the material
		Material<CUDA_RNG>* m;

		if (rng.sample() > .5f) {

			m = new Metal<CUDA_RNG>(color, roughness);

		} else {

			m = new Diffuse<CUDA_RNG>(color);

		}


		scenery[id] = new CUDASphere(center, r, m);
	}

}

CUDAVisible** single_ball()
{

    CUDAVisible** scenery;

	hipMalloc(&scenery, sizeof(CUDAVisible*));

	gen_single_ball << <1, 1>> > (scenery);

	hipDeviceSynchronize();

	return scenery;
}

__global__ void gen_single_ball(CUDAVisible** const scenery)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if (id < 1)
	{
		vec3 center = vec3(3.f, 0.f, 0.f);
		float radius = 1.f;
		Material<CUDA_RNG>* mat = new Diffuse<CUDA_RNG>(vec3(1.f, 0.f, 0.f));
		scenery[id] = new CUDASphere(center, radius, mat);
	}
}