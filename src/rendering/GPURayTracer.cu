#include "hip/hip_runtime.h"
#include "rendering/GPURayTracer.cuh"

#include "rendering/GPURayTracerKernels.cuh"

#define DEBUG false

using std::shared_ptr;
using std::cout;
using std::endl;

void GPURayTracer::terminateRays(UnifiedArray<Ray>* p_rayArray, UnifiedArray<uint32_t>* p_activeRayIndices)
{

	uint32_t threads = max_threads;
	uint32_t blocks = p_activeRayIndices->size() / threads + 1;

	cuda_terminate_rays << <blocks, threads >> > (p_rayArray, p_activeRayIndices);

	checkCudaErrors(hipDeviceSynchronize());
}

GPURayTracer::GPURayTracer(RayTracerConfig config)
	: xRes(config.xRes)
	, yRes(config.yRes)
	, spp(config.spp)
	, maxBounce(config.maxBounce)
	, rayCount(spp * xRes * yRes)
	, raysPerBatch(std::min(rayCount, spp * (maxRaysPerBatch / spp)))
	, ixnEngine(std::make_unique<BranchingTriangleIntersector>(config.minFreePath), config.minFreePath) 
{

	showDeviceProperties();

	increaseStackLimit();
}

shared_ptr<FrameBuffer> GPURayTracer::render(const Scene& scene, const Camera& camera)
{

	// Allocate Frame Array
	shared_ptr<FrameBuffer> m_fb { make_managed<FrameBuffer>(yRes, xRes) };

	// Make a copy of the camera in managed memory
	m_cam =  make_managed<Camera>(camera);

	// Allocate ray data (Ray, colour, rng)
	allocate_rays();

	// RNG for each ray
	create_rngs();

	// Package these in an IntersectionArray
	shared_ptr<UnifiedArray<Intersection>> m_triangleIntersectionArray = make_managed<UnifiedArray<Intersection>>(m_rayArray->size());
	shared_ptr<UnifiedArray<Intersection>> m_sphereIntersectionArray = make_managed<UnifiedArray<Intersection>>(m_rayArray->size());

	auto m_mesh{ scene.m_mesh->getFinder() };

	auto m_rayBundle{ make_managed<RayBundle>(RayBundle{m_rayArray.get(), nullptr, m_triangleIntersectionArray.get(), m_sphereIntersectionArray.get()}) };

	using milli = std::chrono::milliseconds;

	auto start = std::chrono::high_resolution_clock::now();

	for (uint64_t rayIDOffset = 0; rayIDOffset < rayCount; rayIDOffset += raysPerBatch)
	{

		std::cout << "Ray progress " << (float)rayIDOffset / (float)rayCount * 100 << "% " << rayIDOffset << " / " << rayCount << std::endl;

		generatePrimaryRays(rayIDOffset, m_fb.get());

		shared_ptr<UnifiedArray<uint32_t>> m_activeRayIndices{ resetActiveRays(m_rayArray->size()) };


		for (uint16_t bounce = 0; bounce < maxBounce; bounce++)
		{

			cout << endl << "  Bounce " << bounce << endl;

			m_rayBundle->p_activeRayIndices = m_activeRayIndices.get();

			ixnEngine.run(m_rayBundle.get(), m_mesh.get(), scene.m_sphereArray.get());

			colourRays(m_rayArray.get(), m_activeRayIndices.get(), m_mesh->p_triangleColourArray, scene.m_sphereColourArray.get(), m_triangleIntersectionArray.get(), m_sphereIntersectionArray.get());

			m_activeRayIndices = gatherActiveRays(m_activeRayIndices.get(), m_triangleIntersectionArray.get(), m_sphereIntersectionArray.get());

			cout << "  " << m_activeRayIndices->size() << " rays still active" << endl;

			if (m_activeRayIndices->size() == 0)
				break;

			scatterRays(m_rayArray.get(), m_activeRayIndices.get(), m_mesh->p_vertexArray, m_mesh->p_indexArray, scene.m_sphereArray.get(), m_triangleIntersectionArray.get(), m_sphereIntersectionArray.get());

			cout << endl;
		}

		terminateRays(m_rayArray.get(), m_activeRayIndices.get());

		renderRays(rayIDOffset, m_fb.get());

		std::cout << std::endl;
	}

	checkCudaErrors(hipDeviceSynchronize());

	auto finish = std::chrono::high_resolution_clock::now();
	
	std::cout << "Frametime: "
		<< std::chrono::duration_cast<milli>(finish - start).count() << "ms"
		<< std::endl;

	return m_fb;
}

void GPURayTracer::allocate_rays()
{

	// Allocate rays
	m_rayArray = make_managed<UnifiedArray<Ray>>(raysPerBatch);


	checkCudaErrors(hipDeviceSynchronize());

}

void GPURayTracer::create_rngs()
{

	m_rngs = make_managed<UnifiedArray<CUDA_RNG>>(raysPerBatch);

	hipDeviceSynchronize();

	uint32_t threads = max_threads;
	uint32_t blocks = raysPerBatch / threads + 1;

	cuda_create_rngs << <blocks, threads >> > (m_rngs.get());

	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipDeviceSynchronize());
}

void GPURayTracer::generatePrimaryRays(const uint64_t ray_offset_index, const FrameBuffer* const m_fb)
{

	uint32_t threads = max_threads;
	uint32_t blocks = raysPerBatch / threads + 1;

	std::cout << "generatePrimaryRays blocks: " << blocks << ", threads: " << threads << std::endl;

	cuda_gen_rays<<<blocks, threads>>>(m_rayArray->data(), rayCount, raysPerBatch, ray_offset_index, m_cam.get(), m_fb, m_rngs->data(), spp);

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipGetLastError());

}

void GPURayTracer::showDeviceProperties()
{

	cout << "Device Properties" << endl;

	hipDeviceProp_t prop;

	hipGetDeviceProperties(&prop, 0); // assume one CUDA device

	cout << "Max Grid Size: " << prop.maxGridSize[0] << "x " << prop.maxGridSize[1] << "y " << prop.maxGridSize[2] << "z " << endl;
	cout << "Max Threads Per Block: " << prop.maxThreadsPerBlock << endl;
	cout << "Shared Mem Per Block: " << prop.sharedMemPerBlock << endl;

	cout << endl;
}

void GPURayTracer::increaseStackLimit()
{

	size_t stack_limit;

	checkCudaErrors(hipDeviceGetLimit(&stack_limit, hipLimitStackSize));

	std::cout << "Default stack limit: " << stack_limit << std::endl;

	checkCudaErrors(hipDeviceSetLimit(hipLimitStackSize, 8*stack_limit));

	checkCudaErrors(hipDeviceGetLimit(&stack_limit, hipLimitStackSize));

	std::cout << "New stack limit: " << stack_limit << std::endl;

	cout << endl;
}


void GPURayTracer::colourRays(UnifiedArray<Ray>* p_rayArray, UnifiedArray<uint32_t>* p_activeRayIndices, UnifiedArray<vec3>* p_triangleColurArray, UnifiedArray<vec3>* p_sphereColourArray, UnifiedArray<Intersection>* p_triangleIntersectionArray, UnifiedArray<Intersection>* p_sphereIntersectionArray)
{

	uint32_t threads = max_threads;
	uint32_t blocks = raysPerBatch / threads + 1;

	cuda_colour_rays << <blocks, threads >> > (p_rayArray, p_activeRayIndices, p_triangleColurArray, p_sphereColourArray, p_triangleIntersectionArray, p_sphereIntersectionArray);

	checkCudaErrors(hipDeviceSynchronize());
}

void GPURayTracer::renderRays(const uint64_t ray_offset_index, FrameBuffer* m_fb)
{

	uint32_t threads = max_threads;

	uint64_t pixel_start_idx = ray_offset_index / (uint64_t)spp;

	uint64_t pixel_end_idx = pixel_start_idx + raysPerBatch / spp; // not including this index

	pixel_end_idx = std::min(pixel_end_idx, static_cast<uint64_t>(m_fb->h * m_fb->w));

	uint64_t pixel_batch_size = pixel_end_idx - pixel_start_idx;

	uint64_t blocks = pixel_batch_size / threads + 1;

	std::cout << "render_rays blocks: " << blocks << ", threads: " << threads << std::endl;

	cuda_render_rays << <blocks, threads >> > (pixel_start_idx, pixel_end_idx, m_rayArray.get(), m_fb, spp);

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipGetLastError());
}

shared_ptr<UnifiedArray<uint32_t>> GPURayTracer::resetActiveRays(const uint32_t& bufferSize)
{

	shared_ptr<UnifiedArray<uint32_t>> m_activeRayIndices{ make_managed<UnifiedArray<uint32_t>>(bufferSize) };

	uint32_t threads = max_threads;
	uint32_t blocks = raysPerBatch / threads + 1;

	cuda_reset_active_rays << <blocks, threads >> > (m_activeRayIndices.get());

	checkCudaErrors(hipDeviceSynchronize());

	return m_activeRayIndices;
}

std::shared_ptr<UnifiedArray<uint32_t>> GPURayTracer::gatherActiveRays(UnifiedArray<uint32_t>* p_activeRayIndices, UnifiedArray<Intersection>* p_triangleIntersectionArray, UnifiedArray<Intersection>* p_sphereIntersectionArray)
{

	uint32_t length = p_activeRayIndices->size();

	// Create 0, 1 mask for intersections
	UnifiedArray<uint32_t>* p_mask = new UnifiedArray<uint32_t>(length);

	// Create scan of mask
	UnifiedArray<uint32_t>* p_scan = new UnifiedArray<uint32_t>(length);

	KernelLaunchParams klp(max_threads);

	cuda_is_active<<<klp.blocks(length), klp.maxThreads>>>(p_mask, p_triangleIntersectionArray, p_sphereIntersectionArray);
	checkCudaErrors(hipDeviceSynchronize());

	cudaScan(p_mask, p_scan);
	checkCudaErrors(hipDeviceSynchronize());

	uint32_t activeRayCount = (*p_scan)[length - 1];

	if (activeRayCount > length)
		throw std::runtime_error("Scan failed: more active rays than rays\n");

	// Create new active ray index array
	shared_ptr<UnifiedArray<uint32_t>> m_newActiveRayIndices{ make_managed<UnifiedArray<uint32_t>>(activeRayCount) };

	for (uint32_t i = 0; i < length; i++)
	{

		if ((*p_mask)[i] == 1)
			(*m_newActiveRayIndices)[(*p_scan)[i] - 1] = (*p_activeRayIndices)[i];
	}

	delete p_mask;
	delete p_scan;
	
	return m_newActiveRayIndices;
}

void GPURayTracer::scatterRays(UnifiedArray<Ray>* p_rayArray, UnifiedArray<uint32_t>* p_activeRayIndices, UnifiedArray<vec3>* p_vertexArray, UnifiedArray<uint32_t>* p_indexArray, UnifiedArray<CUDASphere>* p_sphereArray, UnifiedArray<Intersection>* p_triangleIntersectionArray, UnifiedArray<Intersection>* p_sphereIntersectionArray)
{

	KernelLaunchParams klp(max_threads);

	cuda_scatter_rays << <klp.blocks(p_activeRayIndices->size()), klp.maxThreads >> > (
		p_rayArray,
		p_activeRayIndices,
		p_vertexArray,
		p_indexArray,
		p_sphereArray,
		p_triangleIntersectionArray,
		p_sphereIntersectionArray,
		m_rngs->data()
	);

	checkCudaErrors(hipDeviceSynchronize());
}


/*

void GPURayTracer::shade_rays(const uint64_t ray_offset_index)
{
	
	uint32_t threads = 512;

	uint32_t blocks = raysPerBatch / threads + 1;

	std::cout << "shade_rays blocks: " << blocks << ", threads: " << threads << std::endl;

	size_t stack_size;

	checkCudaErrors(cudaThreadGetLimit(&stack_size, hipLimitStackSize));

	cuda_shade_ray << <blocks, threads >> > (rays, ray_colours, rayCount, raysPerBatch, ray_offset_index, visibles, maxBounce, minFreePath, rngs);

	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipGetLastError());
}


*/
